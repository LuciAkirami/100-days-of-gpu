
#include <hip/hip_runtime.h>
#include<stdio.h> // Include standard I/O library for printing

const int N = 1 << 20;

__global__ void kernel(float *x, int n)
{
    int tid = threadIdx.x + blockIdx.x * blockDim.x;
    for (int i = tid; i < n; i += blockDim.x * gridDim.x) {
        x[i] = sqrt(pow(3.14159,i));
    }
}

int main()
{
    const int num_streams = 8;
    float ms;

    hipEvent_t startEvent, stopEvent;

    hipEventCreate(&startEvent);
    hipEventCreate(&stopEvent);
    
    hipStream_t streams[num_streams];
    float *data[num_streams];

    hipEventRecord(startEvent, 0);
    for (int i = 0; i < num_streams; i++) {
        hipStreamCreate(&streams[i]);
 
        hipMalloc(&data[i], N * sizeof(float));
        
        // launch one worker kernel per stream
        kernel<<<1, 64, 0, streams[i]>>>(data[i], N);

        // launch a dummy kernel on the default stream
        kernel<<<1, 1>>>(0, 0);
    }
    hipEventRecord(stopEvent, 0);
    hipEventSynchronize(stopEvent);

    hipEventElapsedTime(&ms, startEvent, stopEvent);

    printf("Elapsed Time: %f\n",ms);
    hipDeviceReset();

    return 0;
}