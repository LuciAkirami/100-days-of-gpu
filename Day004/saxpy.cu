#include "hip/hip_runtime.h"
#include <stdio.h>

__global__
void saxpy(float *x, float *y, float a, int N){
    int idx = blockIdx.x * blockDim.x + threadIdx.x;

    if(idx < N){
        y[idx] = a * x[idx] + y[idx];
    }
}

int main(){
    float *x, *y, *x_d, *y_d;
    int N = 1 << 20; // 1 Million
    int size = N * sizeof(float);

    x = (float *)malloc(size);
    y = (float *)malloc(size);

    for(int i=0; i<N; i++){
        x[i] = 1.0f;
        y[i] = 2.0f;
    }

    int numThreadsPerBlock = 256;
    int numBlocks = ceil(N / numThreadsPerBlock);

    hipMalloc((void **)&x_d, size);
    hipMalloc((void **)&y_d, size);

    hipMemcpy(x_d, x, size, hipMemcpyHostToDevice);
    hipMemcpy(y_d, y, size, hipMemcpyHostToDevice);

    saxpy<<<numBlocks, numThreadsPerBlock>>>(x_d, y_d, 2.0f, N);

    hipMemcpy(y, y_d, size, hipMemcpyDeviceToHost);

    float maxError = 0.0f;
    for(int i=0; i<N; i++){
        maxError = max(maxError, abs(y[i] - 4.0f));
    }

    printf("Max Error is %.2f\n",maxError);

    hipFree(x_d);
    hipFree(y_d);

    free(x);
    free(y);
}